#include "hip/hip_runtime.h"
//
//  main.c
//  Fits2Movie
//
//  Created by Antoine Genetelli on 27/05/14.
//  Copyright (c) 2014 Antoine Genetelli. All rights reserved.
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <unistd.h>
#include <fitsio.h>
#include <stdint.h>
#ifndef __APPLE__
#include <argp.h>
#endif
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "kernelConv.cuh"

extern "C" {

#ifndef __APPLE__
// Argp
const char *argp_program_version ="Fits2Movie 0.1";
const char *argp_program_bug_address ="<antoine.genetelli@mac.com>";
#endif

#include "aviFunction.h"
#include "fitsFunction.h"
#include "parserCmdLine.h"
}

int main(int argc, char * argv[]){
	printf("Welcome to %s!\n",argv[0]);
	struct arguments arguments;
	arguments.scale=0;
	arguments.fps=30;
	printf("FPS = %i\n",arguments.fps);
	int error=0;
#ifndef __APPLE__
	error = argp_parse (&argp, argc, argv, 0, 0, &arguments);
#else
	error = parseCmdLine(argc,argv,optString,&arguments);
#endif
	printf("error = %i\n",error);
	if (error != 0){
		exit(-1);
	}
	printf("Number of files = %i\n",argc);
	printf("Save movie in : %s\n",arguments.output);
	printf("First fit files = %s\n",argv[arguments.itStart]);
	printf("FPS = %i\n",arguments.fps);

	double dmin=arguments.dMinMax[0];
	double dmax=arguments.dMinMax[1];

	printf("Scaling parameters : %lf,%lf",dmin,dmax);

	// Cuda
	int nbCuda=0;
	nbCuda=checkCudaDevice();
	printf("There is %i devices\n",nbCuda);

	// Fits Variables
	int status=0;
	int imgSize[]={0,0,0};
	int min=0,max=0;

	// Get image dimension
	status=getImageSize(argv[arguments.itStart],imgSize,&min,&max);
	if (status != 0) {
		fits_report_error(stderr,status);
		exit(-1);
	}

	// AVCodec variable
	printf("AV struct \n");
	struct AVFormatContext *oc;
	struct AVCodec *avCodec;
	struct AVStream *avStream;
	struct AVFrame *frameRGB,*frameYUV;
	struct AVFrame *frameYUVConv;


	// Alloc the frameBuffer for encoding
	size_t bRGB=3*imgSize[1]*imgSize[2]*sizeof(uint8_t);
	size_t bYUV=2*imgSize[1]*imgSize[2]*sizeof(uint8_t);
	size_t bYUVConv=0;
	if (arguments.scale == 1) {
		bYUVConv=bYUV;
		bYUV=2*arguments.NXNY[0]*arguments.NXNY[1]*sizeof(uint8_t);
	}
	uint8_t *hbRGB,*hbYUV,*hbYUVConv;
	hbRGB=(uint8_t *)malloc(bRGB);
	hbYUV=(uint8_t *)malloc(bYUV);
	if (arguments.scale == 1){
		hbYUVConv = (uint8_t *)malloc(bYUVConv);
	}

	// Init avcodec
	av_register_all();
	av_log_set_level(AV_LOG_INFO);

	// Open Movie file and alloc necessary stuff
	remove(arguments.output);
	openFormat(arguments.output, &oc);
	if (arguments.scale == 1){
		openStream(oc, &avCodec, &avStream, arguments.NXNY[0], arguments.NXNY[1], arguments.fps);
		openCodec(&avCodec, avStream);
		allocFrames(avStream, &frameRGB, &frameYUV, hbRGB, hbYUV, imgSize[1], imgSize[2]);
		allocFrameConversion(&frameYUVConv,hbYUVConv,arguments.NXNY[0],arguments.NXNY[1]);
	} else {
		openStream(oc, &avCodec, &avStream, imgSize[1], imgSize[2], arguments.fps);
		openCodec(&avCodec, avStream);
		allocFrames(avStream, &frameRGB, &frameYUV, hbRGB, hbYUV, imgSize[1], imgSize[2]);
	}
	writeHeader(arguments.output, oc);
	printf("Using %s: %s\nCodec: %s\n",oc->oformat->name,oc->oformat->long_name,avcodec_get_name(oc->oformat->video_codec));

	// Alloc buffer fits
	void *data=NULL;
	size_t sData=0;
	sData=allocDataType(&data,imgSize[0],imgSize[1],imgSize[2]);

	// Test if cuda works
	printf("buffer size= %zu, data size = %zu\n",bRGB,sData);
	uint8_t *dbRGB;
	hipMalloc((void **)&dbRGB,bRGB);
	void *dData;
	hipMalloc((void **)&dData, sData);

	for (int i=arguments.itStart; i<argc; i++) {
		printf("Fits: %s\n",argv[i]);
		status=readFits(argv[i],data, imgSize,&min,&max);
		printf("data[min,max]=[%lf,%lf]\n",dmin,dmax);

		// copy data to device
		hipMemcpy(dData, data, sData, hipMemcpyHostToDevice);
		check_CUDA_error("Copying H to D");

		// launch the process
		launchConvertion(dbRGB, dData, imgSize[0], imgSize[1], imgSize[2], dmin, dmax);

		// copy back buffRGB to host
		hipMemcpy(hbRGB,dbRGB,bRGB,hipMemcpyDeviceToHost);
		check_CUDA_error("Copying D to H");

		// Rescale and encode frame
		if (arguments.scale == 1){
			rescaleRGBToYUV(frameRGB,frameYUVConv,hbRGB,hbYUVConv,bRGB);
			rescaleYUV(frameYUVConv,frameYUV,hbYUVConv,hbYUV,bYUV);
		} else {
			rescaleRGBToYUV(frameRGB,frameYUV,hbRGB,hbYUV,bRGB);
		}
		encodeOneFrameYUV(oc,avStream,frameYUV,i);
	}
	hipFree(dbRGB);
	hipFree(dData);
	free(data);

	// dealloc movie files
	av_write_trailer(oc);
	avcodec_close(avStream->codec);
	av_free(avStream);
	avio_close(oc->pb);
	deallocFrames(frameRGB, frameYUV, hbRGB, hbYUV);
	if (arguments.scale == 1) deallocFrameConversion(frameYUVConv,hbYUVConv);

	return 0;
}

