#include "hip/hip_runtime.h"
#include "kernelConv.cuh"

__global__ void convert_fits_RGB(uint8_t *buff, double *data, int nx, int ny, double minD, double maxD){
	int x=blockDim.x*blockIdx.x+threadIdx.x;
	int y=blockDim.y*blockIdx.y+threadIdx.y;
	int CC=y*nx+x;
	double temp=0;
//	printf("data[%i]=%f\n",CC,data[CC]);

	if (CC < nx*ny){
		if (data[CC] < minD) {
			temp =0;
		} else if (data[CC] > maxD){
			temp =255;
		} else {
			temp = (data[CC]-minD)/(maxD-minD)*255;
		}
		// attempt to emule loadct data
		buff[3*CC]=(uint8_t)((1.44068*temp > 255) ? 255 : 1.44068*temp);
		buff[3*CC+1]=(uint8_t)(temp);
		buff[3*CC+2]=(uint8_t)((temp <= 190) ? 0 : 3.92308*temp);
	}
}
void launchConvertion(uint8_t *buff, void *data, int nx, int ny, double minD, double maxD){
	dim3 dimB(BLOCKX,BLOCKY);
	dim3 dimG(nx/BLOCKX,ny/BLOCKY);

	printf("%i,%i\n",nx,ny);
	printf("%i,%i\n",nx/32,ny/32);

	printf("Scaling = %lf,%lf\n",minD,minD);

	// lauch kernel
	convert_fits_RGB<<<dimG,dimB>>>(buff,(double *)data,nx,ny,minD,maxD);
	hipDeviceSynchronize();
	check_CUDA_error("Convertion");
}

// Cuda handling
void check_CUDA_error(const char *message){
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess) {
		printf("ERROR: %s: %s\n", message, hipGetErrorString(error) );
		exit(-1);
	}
}
int checkCudaDevice(){
	int NBCudaDev=0;
	int devN=0;
	hipGetDeviceCount(&NBCudaDev);
	if (NBCudaDev > 1){
		devN=0;
	}

	hipSetDevice(devN);
	hipGetDevice(&devN);
	hipDeviceReset();

	hipSetDeviceFlags(hipDeviceMapHost);
	printf("\nThere is %i CUDA Device using %i",NBCudaDev,devN);
	hipDeviceProp_t dprop;
	hipGetDeviceProperties(&dprop, devN);
	printf(" : %s\n", dprop.name);
	printf("Can Map host Mem : %i\n", dprop.canMapHostMemory);

	// Show device properties
	printf("Max Treads by block = %i.\n",dprop.maxThreadsPerBlock);
	printf("Max Grid Size X = %i.\n\n",dprop.maxGridSize[1]);
	return NBCudaDev;
}
