#include "hip/hip_runtime.h"
#include "kernelConv.cuh"

__global__ void convert_fits_RGB(uint8_t *buff, double *data, int nx, int ny, double minD, double maxD){
	int x=blockDim.x*blockIdx.x+threadIdx.x;
	int y=blockDim.y*blockIdx.y+threadIdx.y;
	int CC=y*nx+x;
	double temp=0;
	printf("data[%i]=%f\n",CC,data[CC]);

	if (CC < nx*ny){
		temp = (data[CC]-minD)/(maxD-minD)*255;
		buff[3*CC]=(1.44068*temp > 255) ? 1.44068*temp : 255;
		buff[3*CC+1]=temp;
		buff[3*CC+2]=(temp <= 190) ? 0 : 3.92308*temp;
	}
}
void launchConvertion(uint8_t *buff, void *data, int nx, int ny, int minD, int maxD){
	dim3 dimB(BLOCKX,BLOCKY);
	dim3 dimG(nx/BLOCKX,ny/BLOCKY);

	printf("%i,%i\n",nx,ny);
	printf("%i,%i\n",nx/32,ny/32);

	double minDD=minD;
	double maxDD=maxD;
	printf("%i,%i\n",minD,minD);
	printf("%f,%f\n",minDD,maxDD);

	// lauch kernel
	convert_fits_RGB<<<dimG,dimB>>>(buff,(double *)data,nx,ny,minDD,maxDD);
	hipDeviceSynchronize();
	check_CUDA_error("Convertion");
}

// Cuda handling
void check_CUDA_error(const char *message){
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess) {
		printf("ERROR: %s: %s\n", message, hipGetErrorString(error) );
		exit(-1);
	}
}
int checkCudaDevice(){
	int NBCudaDev=0;
	int devN=0;
	hipGetDeviceCount(&NBCudaDev);

	hipSetDevice(devN);
	hipGetDevice(&devN);
	hipDeviceReset();

	hipSetDeviceFlags(hipDeviceMapHost);
	printf("\nThere is %i CUDA Device using %i",NBCudaDev,devN);
	hipDeviceProp_t dprop;
	hipGetDeviceProperties(&dprop, devN);
	printf(" : %s\n", dprop.name);
	printf("Can Map host Mem : %i\n", dprop.canMapHostMemory);

	// Show device properties
	printf("Max Treads by block = %i.\n",dprop.maxThreadsPerBlock);
	printf("Max Grid Size X = %i.\n\n",dprop.maxGridSize[1]);
	return NBCudaDev;
}
